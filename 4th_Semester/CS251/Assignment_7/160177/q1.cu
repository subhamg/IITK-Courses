
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define ERROR(s) printf("%s \n Usage: %s <no. of elements> <no. of rows> <no. of columns>\n", s, argv[0]); exit(-1);

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array
{
                    double num1;
                    double num2;
                    double result;
};



__device__ void function(struct num_array *a)
{
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    return;
}
__global__ void calculate(char *mem, int num)
{
      int x = blockDim.x * blockIdx.x + threadIdx.x;
      int y = threadIdx.y;
      int i = x + y * blockDim.x * gridDim.x;
      if(i >= num)
           return;
       struct num_array *a = (struct num_array *)(mem + (i * 3 * sizeof(double)));
      function(a);
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks;
    int rows, cols, threads;

    if(argc != 4){
        ERROR("Invalid number of parameters!");
    }

    num = atoi(argv[1]);
    if(num<=0)
    {
        ERROR("Invalid number of elements!");
    }

    rows = atoi(argv[2]);
    if(rows<=0)
    {
        ERROR("Invalid number of rows");
    }

    cols = atoi(argv[3]);
    if(cols<=0)
    {
        ERROR("Invalid number of columns");
    }

    if(rows * cols > 1024 || rows * cols <= 0)
    {
        ERROR("rows * cols <= 1024");
    }
    /* Allocate host (CPU) memory and initialize*/

    ptr = (char *)malloc(num * 3 * sizeof(double));
    sptr = ptr;
    for(i=0; i<num; ++i){
       pa = (struct num_array *) sptr;
       pa->num1 = (double) i + (double) i * 0.1;
       pa->num2 = pa->num1 + 1.0;
       sptr += 3 * sizeof(double);
    }
    
    threads = rows * cols;

    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = num / threads;
    
    dim3 grid(rows, cols);

    if(num % threads)
           ++blocks;

    calculate<<<blocks, grid>>>(gpu_mem, num);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
//    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;
   
    /*Print the last element for sanity check*/ 
    pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("num1=%f num2=%f result=%f\n", pa->num1, pa->num2, pa->result);

    
    free(ptr);
}
